#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>

using std::vector;

#define BLOCK_SIZE 1024

__global__ void powKernel(double *dev_vect, int N) {
  // Глобольный тред id
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    printf("test: %i",tid);
    if (tid < N) {
        dev_vect[tid] = pow(dev_vect[tid], dev_vect[tid]);
    }
}

int main() {
    int N;
    std::cin>>N;

    size_t bytes = sizeof(double) * N;

    vector<double> host_vect(N);
    vector<double> host_vect_out(N);
    double num;

    while (std::cin >> num) {
        // add number to vector
        host_vect.push_back(num);
    }

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    double *dev_vect;
    hipMalloc(&dev_vect, bytes);

    hipMemcpy(dev_vect, host_vect.data(), bytes, hipMemcpyHostToDevice);


    hipEvent_t start,stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    //run kernel
    powKernel<<<num_blocks, BLOCK_SIZE>>>(dev_vect, N);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    // Уничтожаем созданные эвенты
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(host_vect_out.data(), dev_vect, bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i<N; i++)
        std::cout<<host_vect_out[i]<<" ";
    std::cout<<std::endl;

    hipFree(dev_vect);

    return 0;
}