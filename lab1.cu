
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>

using std::vector;

#define BLOCK_SIZE 1024

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                                        \
 hipError_t e=hipGetLastError();                                                \
 if(e!=hipSuccess) {                                                             \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));  \
   exit(0);                                                                       \
 }                                                                                \
}

__global__ void powKernel(double *dev_vect, int N) {
  // Глобольный тред id
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid < N) {
        dev_vect[tid] = dev_vect[tid] * dev_vect[tid];
    }
}

int main() {
    int N;
    std::cin>>N;

    if (N>pow(2,25)){
        fprintf(stderr,"ERROR: N is too big");
        return 0;
    }

    size_t bytes = sizeof(double) * N;

    vector<double> host_vect;
    vector<double> host_vect_out(N);
    double num;

    while (std::cin >> num) {
        // add number to vector
        host_vect.push_back(num);
    }

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    double *dev_vect;
    hipMalloc(&dev_vect, bytes);

    hipMemcpy(dev_vect, host_vect.data(), bytes, hipMemcpyHostToDevice);
    cudaCheckError();


    hipEvent_t start,stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    //run kernel
    powKernel<<<num_blocks, BLOCK_SIZE>>>(dev_vect, N);
    cudaCheckError();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    // Уничтожаем созданные эвенты
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(host_vect_out.data(), dev_vect, bytes, hipMemcpyDeviceToHost);
    cudaCheckError();

    for (int i = 0; i<N; i++)
        printf("%.10e ",host_vect_out[i]);

    hipFree(dev_vect);

    return 0;
}